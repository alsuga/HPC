#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include <bits/stdc++.h>

#define TILE_WIDTH 32

using namespace std;

//matrix initialization
void init(int *A,int n, int d){
  for(int i = 0; i < n*n; i++)
    A[i] = d;
}

//matrix comparation
bool compare(int *A, int *B, int rows, int cols){
  for(int i = 0; i < rows*cols; i++)
    if(A[i] != B[i])
      return false;
  return true;
}

//print matrix
void printmat(int *A, int rows, int cols){
  for(int i = 0; i < rows; i++){
    for(int j = 0; j < cols; j++){
      cout<<A[i*rows+j]<<" ";
    }
    cout<<endl;
  }
  cout<<endl;
}

//matrix multiplication
void matMult(int *h_A, int *h_B, int *h_C, int common, int Arows, int Bcols){
  int sum;
  for(int i = 0; i < Arows; i++)
    for(int j = 0; j < Bcols; j++){
      sum = 0;
      for(int k = 0; k < common; k++)
        sum += h_A[common*i + k] * h_B[Bcols*k + j];
      h_C[Bcols*i + j] = sum;
       // h_C[n*i + j] += h_A[n*i + k] * h_B[n*k + j];
    }
}


//Parallel kernel
__global__ void matMultPP (int *A, int *B, int *C, int n){
  int i = threadIdx.y + blockDim.y * blockIdx.y;
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  if(i < n and j < n){
    int sum = 0;
    for(int k = 0; k < n; ++k)
      sum += A[n*i + k] * B[n*k + j];
    C[n*i + j] = sum;
  }
}

//Parallel kernel (tiling)
__global__ void matrixMulKernelTiled(int *d_M, int *d_N, int *d_P, int width){
  __shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ int Nds[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;
  float Pvalue = 0;
  for(int m = 0; m < width / TILE_WIDTH; ++m){
    Mds[ty][tx] = d_M[row*width + m*TILE_WIDTH + tx];
    Nds[ty][tx] = d_N[(m*TILE_WIDTH + ty) * width + col];
    __syncthreads();
    for(int k = 0; k < TILE_WIDTH; ++k){
      Pvalue += Mds[ty][k] * Nds[k][tx];
    }
    __syncthreads();
  }
  d_P[row*width+col] = Pvalue;
}

int main(){
  int n; cin>>n;
  cout<<n<<endl;
  int size = n*n*sizeof(int);
  int *A = (int *)malloc(size);
  int *B = (int *)malloc(size);
  int *C = (int *)malloc(size);
  int *D = (int *)malloc(size);
  int *d_A, *d_B, *d_C;
  init(A,n,1);
  init(B,n,2);
  init(C,n,0);
  init(D,n,0);
  double a, b;
  clock_t t = clock();

  //Secuencial
  matMult(A,B,C,n);
  t = clock() - t;
  a = ((float)t)/CLOCKS_PER_SEC;
  cout<<a<<endl;
  int block_size = 32;

  //paralelo
  t = clock();

  //Allocate memory for device
  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);
  //Copy Data from host to device
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
  //Blocks and Grids

  dim3 dimBlock(block_size,block_size);
  dim3 dimGrid(ceil(n/(float)block_size),ceil(n/(float)block_size));

  //Launch Kernel
  matMultPP<<<dimGrid, dimBlock>>> (d_A, d_B, d_C, n);
  hipDeviceSynchronize();
  //Copy from device, free device memory
  hipMemcpy (D, d_C, size, hipMemcpyDeviceToHost);


  //matMultP(A,B,D,size);
  t = clock() - t;
  b = ((float)t)/CLOCKS_PER_SEC;
  cout<<b<<endl;
  cout<<(a/b)<<endl;
  //printmat(C,n);
  //printmat(D,n);

  //if(compare(C,D,n)) cout<<"Work :)"<<endl;
  //else cout<<"No work :("<<endl;

  free(A);
  free(B);
  free(C);
  free(D);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  return 0;
}
