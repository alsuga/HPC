#include "hip/hip_runtime.h"
#include <cv.h>
#include <hip/hip_runtime.h>
#include <highgui.h>
#include <bits/stdc++.h>

#define RED 2
#define GREEN 1
#define BLUE 0
#define MASK_WIDTH 3
#define BLOCK_SIZE 32
#define gpu_error(ans) { gpu_assert((ans), __LINE__); }

using namespace cv;
using namespace std;


__constant__ int d_maskc[MASK_WIDTH * MASK_WIDTH];

inline void gpu_assert(hipError_t code, int line){
    if (code != hipSuccess)
          cerr<<"GPUerror: "<<hipGetErrorString(code)<<" in "<< line<<endl;
}

typedef unsigned char uchar;

__host__ __device__
uchar sol(int i, int j) {
  i = (i < 0)? 0 : i;
  i = (i > 254)? 254 : i;
  j = (j < 0)? 0 : j;
  j = (j > 255)? 255 : j;

  int out = sqrt((double)(i*i + j*j));
  return (out > 255)? 255 : out;
}

__global__
void D_grisesN(uchar *rgbImage, uchar *grayImage, int width, int height) {
  size_t i = blockIdx.y*blockDim.y+threadIdx.y;
  size_t j = blockIdx.x*blockDim.x+threadIdx.x;
  if((i < height) && (j < width))
  grayImage[i*width + j] = rgbImage[(i*width + j)*3 + RED] * 0.299 + rgbImage[(i*width+ j)*3 + GREEN] * 0.587\
                         + rgbImage[(i*width + j)*3 + BLUE] * 0.114;
}

__host__
void D_grises(uchar *h_rgbImage, uchar *h_grayImage, int width, int height) {
  uchar *d_rgbImage, *d_grayImage;
  int size = sizeof(uchar) * width * height;
  gpu_error(hipMalloc(&d_rgbImage, size * 3 ));
  gpu_error(hipMemcpy(d_rgbImage, h_rgbImage, size * 3, hipMemcpyHostToDevice));
  gpu_error(hipMalloc(&d_grayImage, size));
  dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE,1);
  dim3 dimGrid(ceil(width/float(BLOCK_SIZE)),ceil(height/float(BLOCK_SIZE)),1);
  D_grisesN<<<dimGrid,dimBlock>>>(d_rgbImage, d_grayImage, width, height);
  hipDeviceSynchronize();
  gpu_error(hipMemcpy(h_grayImage, d_grayImage, size, hipMemcpyDeviceToHost) );
}



__host__
void H_grises(uchar *rgbImage, uchar *grayImage, int width, int height) {
  for(int i = 0; i < height; i++) {
    for(int j = 0; j < width; j++){
      grayImage[i*width + j] = rgbImage[(i*width + j)*3 + RED] * 0.299 + rgbImage[(i*width+ j)*3 + GREEN] * 0.587\
                             + rgbImage[(i*width + j)*3 + BLUE] * 0.114;
    }
  }
}

__global__
void D_sobelN(uchar *grayImage, int *mask, uchar *sobelImage, int width, int height) {
  int tmp, s_row, s_col, pv1, pv2;
  size_t i = blockIdx.y*blockDim.y+threadIdx.y;
  size_t j = blockIdx.x*blockDim.x+threadIdx.x;
  if(i < height and j < width) {
    tmp = 0;
    pv1 = pv2 = 0;
    s_row = i - (MASK_WIDTH/2);
    s_col = j - (MASK_WIDTH/2);
    for(int mask_i = 0; mask_i < MASK_WIDTH; mask_i++) {
      for(int mask_j = 0; mask_j < MASK_WIDTH; mask_j++) {
        if(s_row + mask_i >= 0 and s_row + mask_i < height and s_col + mask_j >= 0 and s_col + mask_j < width) {
          tmp =  (int)grayImage[(s_row+mask_i)*width +(s_col+mask_j)];
          pv1 += tmp * mask[mask_i * MASK_WIDTH + mask_j];
          pv2 += tmp * mask[mask_j * MASK_WIDTH + mask_i];
        }
      }
    }
    sobelImage[i*width + j] = sol(pv1, pv2);
  }
}

__global__
void D_sobelC(uchar *grayImage, uchar *sobelImage, int width, int height) {
  int tmp, s_row, s_col, pv1, pv2;
  size_t i = blockIdx.y*blockDim.y+threadIdx.y;
  size_t j = blockIdx.x*blockDim.x+threadIdx.x;
  if(i < height and j < width) {
    tmp = 0;
    pv1 = pv2 = 0;
    s_row = i - (MASK_WIDTH/2);
    s_col = j - (MASK_WIDTH/2);
    for(int mask_i = 0; mask_i < MASK_WIDTH; mask_i++) {
      for(int mask_j = 0; mask_j < MASK_WIDTH; mask_j++) {
        if(s_row + mask_i >= 0 and s_row + mask_i < height and s_col + mask_j >= 0 and s_col + mask_j < width) {
          tmp =  (int)grayImage[(s_row+mask_i)*width +(s_col+mask_j)];
          pv1 += tmp * d_maskc[mask_i * MASK_WIDTH + mask_j];
          pv2 += tmp * d_maskc[mask_j * MASK_WIDTH + mask_i];
        }
      }
    }
    sobelImage[i*width + j] = sol(pv1, pv2);
  }
}

__global__
void D_sobelT(uchar *grayImage, int *mask, uchar *sobelImage, int width, int height) {
  int tmp, s_row, s_col, pv1, pv2;
  int tile_width = BLOCK_SIZE + MASK_WIDTH - 1;
  __shared__ int tile[tile_width][tile_width];
  size_t i = blockIdx.y*blockDim.y+threadIdx.y;
  size_t j = blockIdx.x*blockDim.x+threadIdx.x;
  //size_t ti = threadIdx.x *
  if(i < height and j < width) {
    tile[i+1][j+1] = grayImage[i*width + j];
    tmp = 0;
    pv1 = pv2 = 0;
    s_row = i - (MASK_WIDTH/2);
    s_col = j - (MASK_WIDTH/2);
    for(int mask_i = 0; mask_i < MASK_WIDTH; mask_i++) {
      for(int mask_j = 0; mask_j < MASK_WIDTH; mask_j++) {
        if(s_row + mask_i >= 0 and s_row + mask_i < height and s_col + mask_j >= 0 and s_col + mask_j < width) {
          tmp =  (int)grayImage[(s_row+mask_i)*width +(s_col+mask_j)];
          pv1 += tmp * mask[mask_i * MASK_WIDTH + mask_j];
          pv2 += tmp * mask[mask_j * MASK_WIDTH + mask_i];
        }
      }
    }
    sobelImage[i*width + j] = sol(pv1, pv2);
  }
}

__host__
void D_sobel(uchar *grayImage, int mask[], uchar* sobelImage, int width, int height) {
  uchar *d_grayImage, *d_sobelImage;
  int *d_mask;
  int size = sizeof(uchar) * width * height;
  gpu_error( hipMalloc(&d_grayImage, size)  );
  gpu_error( hipMemcpy(d_grayImage, grayImage, size, hipMemcpyHostToDevice));
  gpu_error( hipMalloc(&d_sobelImage, size) );
  gpu_error( hipMalloc(&d_mask, MASK_WIDTH * MASK_WIDTH * sizeof(int)) );
  gpu_error( hipMemcpy(d_mask, mask, MASK_WIDTH * MASK_WIDTH * sizeof(int), hipMemcpyHostToDevice));
  dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE,1);
  dim3 dimGrid(ceil(width/float(BLOCK_SIZE)),ceil(height/float(BLOCK_SIZE)),1);
  D_sobelN<<<dimGrid,dimBlock>>>(d_grayImage, d_mask, d_sobelImage, width, height);
  hipDeviceSynchronize();
  gpu_error(hipMemcpy(sobelImage, d_sobelImage, size, hipMemcpyDeviceToHost) );

}

__host__
void H_sobel(uchar *grayImage, int mask[], uchar* sobelImage, int width, int height) {
  int tmp, s_row, s_col, pv1, pv2;
  for(int i = 0; i < height; i++) {
    for(int j = 0; j < width; j++) {
      tmp = 0;
      pv1 = pv2 = 0;
      s_row = i - (MASK_WIDTH/2);
      s_col = j - (MASK_WIDTH/2);
      for(int mask_i = 0; mask_i < MASK_WIDTH; mask_i++) {
        for(int mask_j = 0; mask_j < MASK_WIDTH; mask_j++) {
          if(s_row + mask_i >= 0 and s_row + mask_i < height and s_col + mask_j >= 0 and s_col + mask_j < width) {
            tmp =  (int)grayImage[(s_row+mask_i)*width +(s_col+mask_j)];
            pv1 += tmp * mask[mask_i * MASK_WIDTH + mask_j];
            pv2 += tmp * mask[mask_j * MASK_WIDTH + mask_i];
          }
        }
      }
      sobelImage[i*width + j] = sol(pv1, pv2);
    }
  }
}


int main( ) {
  Mat image;
  uchar *dataimage, *grayimage, *sobelimage;
  image = imread( "img1.jpg",1);
  int Mask[] = {-1, 0, 1, -2 , 0, 2, -1 ,0 ,1};
  dataimage = image.data;

  Mat gray_image, sobel_image;

  Size s = image.size();
  int width = s.width;
  int height = s.height;
  cout<<width<<" "<<height<<endl;
  int sizeGray = sizeof(uchar)*width*height;
  grayimage = (uchar *)malloc(sizeGray);
  sobelimage = (uchar *)malloc(sizeGray);
  D_grises(dataimage, grayimage, width, height);
  D_sobel(grayimage, Mask, sobelimage, width, height);
  //H_grises(dataimage, grayimage, width, height);
  //H_sobel(grayimage, Mask, sobelimage, width, height);

  gray_image.create(height, width, CV_8UC1);
  gray_image.data = grayimage;

  imwrite("./Gray_Image.jpg",gray_image);

  sobel_image.create(height, width, CV_8UC1);
  sobel_image.data = sobelimage;

  imwrite("./Sobel_Image.jpg", sobel_image);
   return 0;
}
