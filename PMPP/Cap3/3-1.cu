#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 1024

using namespace std;

__global__ sum(int *d_A, int *d_B, int *d_C, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < n*n)
    d_C[i] = d_A[i] + d_B[i];
}

__global__ sumR(int *d_A, int *d_B, int *d_C, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for(int j = 0; j < n; j++)
    d_C[i*n + j] = d_A[i*n + j] + d_B[i*n + j];
}

__global__ sumC(int *d_A, int *d_B, int *d_C, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for(int j = 0; j < n; j++)
    d_C[j*n + i] = d_A[j*n + i] + d_B[j*n + i];
}


int main(){
  int *h_A, *h_B, *h_C;
  int n = 5, size = sizeof(int) * n*n;
  h_A = (int *)malloc(size);
  h_B = (int *)malloc(size);
  h_C = (int *)malloc(size);

  for(int i = 0; i < n*n; i++) {
    h_A[i] = 1;
    h_B[i] = 2;
  }
  int *d_A, *d_B, *d_C;

  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  sum<<< ceil( (n*n) / (double)BLOCK_SIZE), BLOCK_SIZE >>> (d_A, d_B, d_C, n);
  sumR<<< ceil( n / (double)BLOCK_SIZE), BLOCK_SIZE >>> (d_A, d_B, d_C, n);
  sumC<<< ceil( n / (double)BLOCK_SIZE), BLOCK_SIZE >>> (d_A, d_B, d_C, n);

  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  free(h_A);
  free(h_B);
  free(h_C);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  return 0;
}
