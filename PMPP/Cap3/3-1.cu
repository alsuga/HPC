#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 1024

using namespace std;

__global__ void sum(int *d_A, int *d_B, int *d_C, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  //if(i < n*n)
    d_C[i] = d_A[i] + d_B[i];
}

__global__ void sumR(int *d_A, int *d_B, int *d_C, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for(int j = 0; j < n; j++)
    d_C[i*n + j] = d_A[i*n + j] + d_B[i*n + j];
}

__global__ void sumC(int *d_A, int *d_B, int *d_C, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for(int j = 0; j < n; j++)
    d_C[j*n + i] = d_A[j*n + i] + d_B[j*n + i];
}


int main(){
  int *h_A, *h_B, *h_C;
  int n; cin>>n;
  int size = sizeof(int) * n*n;
  h_A = (int *)malloc(size);
  h_B = (int *)malloc(size);
  h_C = (int *)malloc(size);

  for(int i = 0; i < n*n; i++) {
    h_A[i] = 3;
    h_B[i] = 4;
  }
  int *d_A, *d_B, *d_C;
  clock_t t = clock();
  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  //sum<<< ceil( (n*n) / (double)BLOCK_SIZE), BLOCK_SIZE >>> (d_A, d_B, d_C, n);
  sumR<<< ceil( n / (double)BLOCK_SIZE), BLOCK_SIZE >>> (d_A, d_B, d_C, n);
  //sumC<<< ceil( n / (double)BLOCK_SIZE), BLOCK_SIZE >>> (d_A, d_B, d_C, n);

  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  double a = ((double)(clock()-t))/CLOCKS_PER_SEC;

  cout<< a <<endl;
  //for(int i = 0; i < n; i++) {
    //for(int j = 0; j < n; j++)
      //cout<<h_C[j]<<" ";
    //cout<<endl;
  //}

  free(h_A);
  free(h_B);
  free(h_C);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  return 0;
}
